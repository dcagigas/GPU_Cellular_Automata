#include "hip/hip_runtime.h"
#include "GOL2_ready_for_ppcg_kernel.hu"
__global__ void kernel0(int *grid)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 32769) {
      if (32 * b0 + t0 <= 16384) {
        grid[(32 * b0 + t0) * 16386 + 0] = grid[(32 * b0 + t0) * 16386 + 16384];
      } else if (32 * b0 + t0 >= 16386) {
        grid[(32 * b0 + t0 - 16385) * 16386 + 16385] = grid[(32 * b0 + t0 - 16385) * 16386 + 1];
      }
      if (b0 >= 512)
        grid[0 * 16386 + (32 * b0 + t0 - 16384)] = grid[16384 * 16386 + (32 * b0 + t0 - 16384)];
      if (32 * b0 + t0 <= 16386)
        grid[16385 * 16386 + (32 * b0 + t0 - 1)] = grid[1 * 16386 + (32 * b0 + t0 - 1)];
    }
}
__global__ void kernel1(int *grid, int *newGrid)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    int private_numNeighbors;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 <= 16384; c0 += 8192)
      if (t0 + c0 >= 1 && t0 + c0 <= 16384)
        for (int c1 = ppcg_max(32 * b1 + 8192, 32 * b1 + 8192 * c0); c1 <= ppcg_min(134234112, 8192 * c0 + 270336); c1 += 8192)
          for (int c3 = ppcg_max(t1, ((t1 + c1 + 15) % 16) + 8192 * t0 + 8192 * c0 - c1 + 1); c3 <= ppcg_min(31, 8192 * t0 + 8192 * c0 - c1 + 16384); c3 += 16) {
            private_numNeighbors = (((((((grid[(t0 + c0 + 1) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3)] + grid[(t0 + c0 - 1) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3)]) + grid[(t0 + c0) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3 + 1)]) + grid[(t0 + c0) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3 - 1)]) + grid[(t0 + c0 + 1) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3 + 1)]) + grid[(t0 + c0 - 1) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3 - 1)]) + grid[(t0 + c0 - 1) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3 + 1)]) + grid[(t0 + c0 + 1) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3 - 1)]);
            if ((grid[(t0 + c0) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3)] == 1) && (private_numNeighbors < 2)) {
              newGrid[(t0 + c0) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3)] = 0;
            } else {
              if ((grid[(t0 + c0) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3)] == 1) && ((private_numNeighbors == 2) || (private_numNeighbors == 3))) {
                newGrid[(t0 + c0) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3)] = 1;
              } else {
                if ((grid[(t0 + c0) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3)] == 1) && (private_numNeighbors > 3)) {
                  newGrid[(t0 + c0) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3)] = 0;
                } else {
                  if ((grid[(t0 + c0) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3)] == 0) && (private_numNeighbors == 3)) {
                    newGrid[(t0 + c0) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3)] = 1;
                  } else {
                    newGrid[(t0 + c0) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3)] = grid[(t0 + c0) * 16386 + (-8192 * t0 - 8192 * c0 + c1 + c3)];
                  }
                }
              }
            }
          }
}
