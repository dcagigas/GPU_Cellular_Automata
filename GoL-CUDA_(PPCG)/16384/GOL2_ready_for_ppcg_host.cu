#include <assert.h>
#include <stdio.h>
#include "GOL2_ready_for_ppcg_kernel.hu"
#include <stdio.h>
#include <stdlib.h>
 
#define SRAND_VALUE 1985

#define dim 16384
#define maxIter 1<<10

 
int print_total_alive (int grid[dim+2][dim+2]);

// Add up all neighbors


void make_Iter (int grid[dim+2][dim+2], int newGrid[dim+2][dim+2]) {
	int i,j, numNeighbors;
	{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

	  int *dev_grid;
	  int *dev_newGrid;
	  
	  cudaCheckReturn(hipMalloc((void **) &dev_grid, (size_t)(16386) * (size_t)(16386) * sizeof(int)));
	  cudaCheckReturn(hipMalloc((void **) &dev_newGrid, (size_t)(16385) * (size_t)(16386) * sizeof(int)));
	  
	  {
	  cudaCheckReturn(hipMemcpy(dev_grid, grid, (size_t)(16386) * (size_t)(16386) * sizeof(int), hipMemcpyHostToDevice));
	  #ifdef STENCILBENCH
	  hipDeviceSynchronize();
	  SB_START_INSTRUMENTS;
	  #endif
	  }
	  {
	  cudaCheckReturn(hipMemcpy(dev_newGrid, newGrid, (size_t)(16385) * (size_t)(16386) * sizeof(int), hipMemcpyHostToDevice));
	  #ifdef STENCILBENCH
	  hipDeviceSynchronize();
	  SB_START_INSTRUMENTS;
	  #endif
	  }
	  {
	    dim3 k0_dimBlock(32);
	    dim3 k0_dimGrid(1025);
	    kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_grid);
	    cudaCheckKernel();
	  }
	  
	  {
	    dim3 k1_dimBlock(16, 32);
	    dim3 k1_dimGrid(256, 256);
	    kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_grid, dev_newGrid);
	    cudaCheckKernel();
	  }
	  
	  {
	  #ifdef STENCILBENCH
	  hipDeviceSynchronize();
	  SB_STOP_INSTRUMENTS;
	  #endif
	  cudaCheckReturn(hipMemcpy(grid, dev_grid, (size_t)(16386) * (size_t)(16386) * sizeof(int), hipMemcpyDeviceToHost));
	  }
	  {
	  #ifdef STENCILBENCH
	  hipDeviceSynchronize();
	  SB_STOP_INSTRUMENTS;
	  #endif
	  cudaCheckReturn(hipMemcpy(newGrid, dev_newGrid, (size_t)(16385) * (size_t)(16386) * sizeof(int), hipMemcpyDeviceToHost));
	  }
	  cudaCheckReturn(hipFree(dev_grid));
	  cudaCheckReturn(hipFree(dev_newGrid));
	}
}


int main(int argc, char* argv[])
{
	int iter, i,j;
    int total = 0; 
	int grid[dim+2][dim+2];
	int newGrid[dim+2][dim+2];
	
	
	//printf ("Random - s\n");
    // Assign initial population randomly
    srand(SRAND_VALUE);
    for(i = 1; i<=dim; i++) {
        for(j = 1; j<=dim; j++) {
            grid[i][j] = rand() % 2;
        }
    }

    // Main game loop
    for (iter = 0; iter<maxIter; iter++) {
        // Left-Right columns

		if (iter%2==0) {
			make_Iter (grid, newGrid);
		} else {
			make_Iter (newGrid, grid);
		}
		
    }// End main game loop
		

	if (iter%2==0)
        total = print_total_alive (grid);
    else
        total = print_total_alive (newGrid);
    printf("Total Alive: %d\n", total);

	//
	
    return 0;
}


int print_total_alive (int grid[dim+2][dim+2]) {
	int i,j;
	int total = 0;
	for (i = 1; i<=dim; i++) {
		for (j = 1; j<=dim; j++) {
			total += grid[i][j];
		}
	}
	//printf("Total Alive: %d\n", total);
	return total;
}

// Result in console: "Total Alive: 45224"
	
