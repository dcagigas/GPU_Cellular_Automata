#include "hip/hip_runtime.h"
#include "GOL2_ready_for_ppcg_kernel.hu"
__global__ void kernel0(int *grid)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 16385) {
      if (32 * b0 + t0 <= 8192) {
        grid[(32 * b0 + t0) * 8194 + 0] = grid[(32 * b0 + t0) * 8194 + 8192];
      } else if (32 * b0 + t0 >= 8194) {
        grid[(32 * b0 + t0 - 8193) * 8194 + 8193] = grid[(32 * b0 + t0 - 8193) * 8194 + 1];
      }
      if (b0 >= 256)
        grid[0 * 8194 + (32 * b0 + t0 - 8192)] = grid[8192 * 8194 + (32 * b0 + t0 - 8192)];
      if (32 * b0 + t0 <= 8194)
        grid[8193 * 8194 + (32 * b0 + t0 - 1)] = grid[1 * 8194 + (32 * b0 + t0 - 1)];
    }
}
__global__ void kernel1(int *grid, int *newGrid)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    int private_numNeighbors;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 <= 8192; c0 += 8192)
      if (t0 + c0 >= 1 && t0 + c0 <= 8192)
        for (int c1 = ppcg_max(32 * b1 + 4096 * c0, 32 * ((b1 + 128) % 256) + 4096); c1 <= ppcg_min(33562624, 4096 * c0 + 135168); c1 += 8192)
          for (int c3 = ppcg_max(t1, ((t1 + c1 + 15) % 16) + 4096 * t0 + 4096 * c0 - c1 + 1); c3 <= ppcg_min(31, 4096 * t0 + 4096 * c0 - c1 + 8192); c3 += 16) {
            private_numNeighbors = (((((((grid[(t0 + c0 + 1) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3)] + grid[(t0 + c0 - 1) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3)]) + grid[(t0 + c0) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3 + 1)]) + grid[(t0 + c0) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3 - 1)]) + grid[(t0 + c0 + 1) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3 + 1)]) + grid[(t0 + c0 - 1) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3 - 1)]) + grid[(t0 + c0 - 1) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3 + 1)]) + grid[(t0 + c0 + 1) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3 - 1)]);
            if ((grid[(t0 + c0) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3)] == 1) && (private_numNeighbors < 2)) {
              newGrid[(t0 + c0) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3)] = 0;
            } else {
              if ((grid[(t0 + c0) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3)] == 1) && ((private_numNeighbors == 2) || (private_numNeighbors == 3))) {
                newGrid[(t0 + c0) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3)] = 1;
              } else {
                if ((grid[(t0 + c0) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3)] == 1) && (private_numNeighbors > 3)) {
                  newGrid[(t0 + c0) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3)] = 0;
                } else {
                  if ((grid[(t0 + c0) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3)] == 0) && (private_numNeighbors == 3)) {
                    newGrid[(t0 + c0) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3)] = 1;
                  } else {
                    newGrid[(t0 + c0) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3)] = grid[(t0 + c0) * 8194 + (-4096 * t0 - 4096 * c0 + c1 + c3)];
                  }
                }
              }
            }
          }
}
