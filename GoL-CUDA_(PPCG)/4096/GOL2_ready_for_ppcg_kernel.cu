#include "hip/hip_runtime.h"
#include "GOL2_ready_for_ppcg_kernel.hu"
__global__ void kernel0(int *grid)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 8193) {
      if (32 * b0 + t0 <= 4096) {
        grid[(32 * b0 + t0) * 4098 + 0] = grid[(32 * b0 + t0) * 4098 + 4096];
      } else if (32 * b0 + t0 >= 4098) {
        grid[(32 * b0 + t0 - 4097) * 4098 + 4097] = grid[(32 * b0 + t0 - 4097) * 4098 + 1];
      }
      if (b0 >= 128)
        grid[0 * 4098 + (32 * b0 + t0 - 4096)] = grid[4096 * 4098 + (32 * b0 + t0 - 4096)];
      if (32 * b0 + t0 <= 4098)
        grid[4097 * 4098 + (32 * b0 + t0 - 1)] = grid[1 * 4098 + (32 * b0 + t0 - 1)];
    }
}
__global__ void kernel1(int *grid, int *newGrid)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    int private_numNeighbors;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 4096)
      for (int c1 = ppcg_max(65536 * b0 + 32 * b1, 32 * ((b1 + 192) % 256) + 2048); c1 <= ppcg_min(8392704, 65536 * b0 + 67584); c1 += 8192)
        for (int c3 = ppcg_max(t1, ((t1 + c1 + 15) % 16) + 65536 * b0 + 2048 * t0 - c1 + 1); c3 <= ppcg_min(31, 65536 * b0 + 2048 * t0 - c1 + 4096); c3 += 16) {
          private_numNeighbors = (((((((grid[(32 * b0 + t0 + 1) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3)] + grid[(32 * b0 + t0 - 1) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3)]) + grid[(32 * b0 + t0) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3 + 1)]) + grid[(32 * b0 + t0) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3 - 1)]) + grid[(32 * b0 + t0 + 1) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3 + 1)]) + grid[(32 * b0 + t0 - 1) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3 - 1)]) + grid[(32 * b0 + t0 - 1) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3 + 1)]) + grid[(32 * b0 + t0 + 1) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3 - 1)]);
          if ((grid[(32 * b0 + t0) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3)] == 1) && (private_numNeighbors < 2)) {
            newGrid[(32 * b0 + t0) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3)] = 0;
          } else {
            if ((grid[(32 * b0 + t0) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3)] == 1) && ((private_numNeighbors == 2) || (private_numNeighbors == 3))) {
              newGrid[(32 * b0 + t0) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3)] = 1;
            } else {
              if ((grid[(32 * b0 + t0) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3)] == 1) && (private_numNeighbors > 3)) {
                newGrid[(32 * b0 + t0) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3)] = 0;
              } else {
                if ((grid[(32 * b0 + t0) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3)] == 0) && (private_numNeighbors == 3)) {
                  newGrid[(32 * b0 + t0) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3)] = 1;
                } else {
                  newGrid[(32 * b0 + t0) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3)] = grid[(32 * b0 + t0) * 4098 + (-65536 * b0 - 2048 * t0 + c1 + c3)];
                }
              }
            }
          }
        }
}
