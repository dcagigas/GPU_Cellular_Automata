#include <assert.h>
#include <stdio.h>
#include "GOL2_ready_for_ppcg_kernel.hu"
#include <stdio.h>
#include <stdlib.h>
 
#define SRAND_VALUE 1985

#define dim 512
#define maxIter 1<<10
 
long int print_total_alive (int *h_grid);

// Add up all neighbors


void make_Iter (int *dev_grid, int *dev_newGrid) {
	{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)


	  {
	    dim3 k0_dimBlock(32);
	    dim3 k0_dimGrid(33);
	    kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_grid);
	    cudaCheckKernel();
	  }
	  
	  {
	    dim3 k1_dimBlock(16, 32);
	    dim3 k1_dimGrid(256, 17);
	    kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_grid, dev_newGrid);
	    cudaCheckKernel();
	  }
	  
	}
}


int main(int argc, char* argv[])
{
	int iter, i,j;
    long int total = 0; 
	int *dev_grid;
	int *dev_newGrid;
	int grid[dim+2][dim+2];
	int newGrid[dim+2][dim+2];
	
	
	//printf ("Random - s\n");
    // Assign initial population randomly
    srand(SRAND_VALUE);
    for(i = 1; i<=dim; i++) {
        for(j = 1; j<=dim; j++) {
            grid[i][j] = rand() % 2;
        }
    }

	  cudaCheckReturn(hipMalloc((void **) &dev_grid, (514) * (514) * sizeof(int)));
	  cudaCheckReturn(hipMalloc((void **) &dev_newGrid, (513) * (514) * sizeof(int)));
	  
	  cudaCheckReturn(hipMemcpy(dev_grid, grid, (514) * (514) * sizeof(int), hipMemcpyHostToDevice));


    // Main game loop
    for (iter = 0; iter<maxIter; iter++) {
        // Left-Right columns

		if (iter%2==0) {
			make_Iter (dev_grid, dev_newGrid);
		} else {
			make_Iter (dev_newGrid, dev_grid);
		}
		
    }// End main game loop

    if (iter%2==0) {
        cudaCheckReturn(hipMemcpy(grid, dev_grid, (dim+2) * (dim+2) * sizeof(int), hipMemcpyDeviceToHost));
        total = print_total_alive ((int *)grid);
    } else {
        cudaCheckReturn(hipMemcpy(newGrid, dev_newGrid, (dim+1) * (dim+2) * sizeof(int), hipMemcpyDeviceToHost));
        total = print_total_alive ((int *)newGrid);
    }
	cudaCheckReturn(hipFree(dev_grid));
	cudaCheckReturn(hipFree(dev_newGrid));
	
    printf("Total Alive: %ld\n", total);
	
    return 0;
}


long int print_total_alive (int *h_grid) {
	int i,j;
	long int total = 0;
	for (i = 1; i<=dim; i++) {
		for (j = 1; j<=dim; j++) {
            total += h_grid[i*(dim+2)+j];
		}
	}
	return total;
}

// Result in console: "Total Alive: 11072"
	
