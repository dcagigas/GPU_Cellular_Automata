#include "hip/hip_runtime.h"
#include "GOL2_ready_for_ppcg_kernel.hu"
__global__ void kernel0(int *grid)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 1025) {
      if (32 * b0 + t0 <= 512) {
        grid[(32 * b0 + t0) * 514 + 0] = grid[(32 * b0 + t0) * 514 + 512];
      } else if (32 * b0 + t0 >= 514) {
        grid[(32 * b0 + t0 - 513) * 514 + 513] = grid[(32 * b0 + t0 - 513) * 514 + 1];
      }
      if (b0 >= 16)
        grid[0 * 514 + (32 * b0 + t0 - 512)] = grid[512 * 514 + (32 * b0 + t0 - 512)];
      if (32 * b0 + t0 <= 514)
        grid[513 * 514 + (32 * b0 + t0 - 1)] = grid[1 * 514 + (32 * b0 + t0 - 1)];
    }
}
__global__ void kernel1(int *grid, int *newGrid)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    int private_numNeighbors;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 512)
      for (int c1 = ppcg_max(8192 * b0 + 32 * b1, 32 * ((b1 + 248) % 256) + 256); c1 <= ppcg_min(131584, 8192 * b0 + 8448); c1 += 8192)
        for (int c3 = ppcg_max(t1, ((t1 + c1 + 15) % 16) + 8192 * b0 + 256 * t0 - c1 + 1); c3 <= ppcg_min(31, 8192 * b0 + 256 * t0 - c1 + 512); c3 += 16) {
          private_numNeighbors = (((((((grid[(32 * b0 + t0 + 1) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3)] + grid[(32 * b0 + t0 - 1) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3)]) + grid[(32 * b0 + t0) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3 + 1)]) + grid[(32 * b0 + t0) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3 - 1)]) + grid[(32 * b0 + t0 + 1) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3 + 1)]) + grid[(32 * b0 + t0 - 1) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3 - 1)]) + grid[(32 * b0 + t0 - 1) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3 + 1)]) + grid[(32 * b0 + t0 + 1) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3 - 1)]);
          if ((grid[(32 * b0 + t0) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3)] == 1) && (private_numNeighbors < 2)) {
            newGrid[(32 * b0 + t0) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3)] = 0;
          } else {
            if ((grid[(32 * b0 + t0) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3)] == 1) && ((private_numNeighbors == 2) || (private_numNeighbors == 3))) {
              newGrid[(32 * b0 + t0) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3)] = 1;
            } else {
              if ((grid[(32 * b0 + t0) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3)] == 1) && (private_numNeighbors > 3)) {
                newGrid[(32 * b0 + t0) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3)] = 0;
              } else {
                if ((grid[(32 * b0 + t0) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3)] == 0) && (private_numNeighbors == 3)) {
                  newGrid[(32 * b0 + t0) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3)] = 1;
                } else {
                  newGrid[(32 * b0 + t0) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3)] = grid[(32 * b0 + t0) * 514 + (-8192 * b0 - 256 * t0 + c1 + c3)];
                }
              }
            }
          }
        }
}
