#include "hip/hip_runtime.h"
#include "GOL2_ready_for_ppcg_kernel.hu"
__global__ void kernel0(int *grid)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 513) {
      if (32 * b0 + t0 <= 256) {
        grid[(32 * b0 + t0) * 258 + 0] = grid[(32 * b0 + t0) * 258 + 256];
      } else if (32 * b0 + t0 >= 258) {
        grid[(32 * b0 + t0 - 257) * 258 + 257] = grid[(32 * b0 + t0 - 257) * 258 + 1];
      }
      if (b0 >= 8)
        grid[0 * 258 + (32 * b0 + t0 - 256)] = grid[256 * 258 + (32 * b0 + t0 - 256)];
      if (32 * b0 + t0 <= 258)
        grid[257 * 258 + (32 * b0 + t0 - 1)] = grid[1 * 258 + (32 * b0 + t0 - 1)];
    }
}
__global__ void kernel1(int *grid, int *newGrid)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    int private_numNeighbors;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 256 && b1 + 256 * ppcg_fdiv_q(128 * b0 - b1 - 1, 256) >= -252 && b1 + 256 * ppcg_fdiv_q(128 * b0 - b1 - 1, 256) <= 776)
      for (int c3 = ppcg_max(t1, ((t1 + 15) % 16) + 4096 * b0 - 32 * b1 + 128 * t0 - 8192 * ppcg_fdiv_q(128 * b0 - b1 - 1, 256) - 8191); c3 <= ppcg_min(31, 4096 * b0 - 32 * b1 + 128 * t0 - 8192 * ppcg_fdiv_q(128 * b0 - b1 - 1, 256) - 7936); c3 += 16) {
        private_numNeighbors = (((((((grid[(32 * b0 + t0 + 1) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3)] + grid[(32 * b0 + t0 - 1) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3)]) + grid[(32 * b0 + t0) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3 + 1)]) + grid[(32 * b0 + t0) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3 - 1)]) + grid[(32 * b0 + t0 + 1) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3 + 1)]) + grid[(32 * b0 + t0 - 1) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3 - 1)]) + grid[(32 * b0 + t0 - 1) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3 + 1)]) + grid[(32 * b0 + t0 + 1) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3 - 1)]);
        if ((grid[(32 * b0 + t0) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3)] == 1) && (private_numNeighbors < 2)) {
          newGrid[(32 * b0 + t0) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3)] = 0;
        } else {
          if ((grid[(32 * b0 + t0) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3)] == 1) && ((private_numNeighbors == 2) || (private_numNeighbors == 3))) {
            newGrid[(32 * b0 + t0) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3)] = 1;
          } else {
            if ((grid[(32 * b0 + t0) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3)] == 1) && (private_numNeighbors > 3)) {
              newGrid[(32 * b0 + t0) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3)] = 0;
            } else {
              if ((grid[(32 * b0 + t0) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3)] == 0) && (private_numNeighbors == 3)) {
                newGrid[(32 * b0 + t0) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3)] = 1;
              } else {
                newGrid[(32 * b0 + t0) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3)] = grid[(32 * b0 + t0) * 258 + (32 * ((128 * b0 + b1) % 256) - 128 * t0 + c3)];
              }
            }
          }
        }
      }
}
