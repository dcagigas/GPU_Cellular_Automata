#include "hip/hip_runtime.h"
#include "GOL2_ready_for_ppcg_kernel.hu"
__global__ void kernel0(int *grid)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 2049) {
      if (32 * b0 + t0 <= 1024) {
        grid[(32 * b0 + t0) * 1026 + 0] = grid[(32 * b0 + t0) * 1026 + 1024];
      } else if (32 * b0 + t0 >= 1026) {
        grid[(32 * b0 + t0 - 1025) * 1026 + 1025] = grid[(32 * b0 + t0 - 1025) * 1026 + 1];
      }
      if (b0 >= 32)
        grid[0 * 1026 + (32 * b0 + t0 - 1024)] = grid[1024 * 1026 + (32 * b0 + t0 - 1024)];
      if (32 * b0 + t0 <= 1026)
        grid[1025 * 1026 + (32 * b0 + t0 - 1)] = grid[1 * 1026 + (32 * b0 + t0 - 1)];
    }
}
__global__ void kernel1(int *grid, int *newGrid)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    int private_numNeighbors;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 1024)
      for (int c1 = ppcg_max(16384 * b0 + 32 * b1, 32 * ((b1 + 240) % 256) + 512); c1 <= ppcg_min(525312, 16384 * b0 + 16896); c1 += 8192)
        for (int c3 = ppcg_max(t1, ((t1 + c1 + 15) % 16) + 16384 * b0 + 512 * t0 - c1 + 1); c3 <= ppcg_min(31, 16384 * b0 + 512 * t0 - c1 + 1024); c3 += 16) {
          private_numNeighbors = (((((((grid[(32 * b0 + t0 + 1) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3)] + grid[(32 * b0 + t0 - 1) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3)]) + grid[(32 * b0 + t0) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3 + 1)]) + grid[(32 * b0 + t0) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3 - 1)]) + grid[(32 * b0 + t0 + 1) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3 + 1)]) + grid[(32 * b0 + t0 - 1) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3 - 1)]) + grid[(32 * b0 + t0 - 1) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3 + 1)]) + grid[(32 * b0 + t0 + 1) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3 - 1)]);
          if ((grid[(32 * b0 + t0) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3)] == 1) && (private_numNeighbors < 2)) {
            newGrid[(32 * b0 + t0) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3)] = 0;
          } else {
            if ((grid[(32 * b0 + t0) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3)] == 1) && ((private_numNeighbors == 2) || (private_numNeighbors == 3))) {
              newGrid[(32 * b0 + t0) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3)] = 1;
            } else {
              if ((grid[(32 * b0 + t0) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3)] == 1) && (private_numNeighbors > 3)) {
                newGrid[(32 * b0 + t0) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3)] = 0;
              } else {
                if ((grid[(32 * b0 + t0) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3)] == 0) && (private_numNeighbors == 3)) {
                  newGrid[(32 * b0 + t0) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3)] = 1;
                } else {
                  newGrid[(32 * b0 + t0) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3)] = grid[(32 * b0 + t0) * 1026 + (-16384 * b0 - 512 * t0 + c1 + c3)];
                }
              }
            }
          }
        }
}
