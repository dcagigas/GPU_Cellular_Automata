#include "hip/hip_runtime.h"
#include "GOL2_ready_for_ppcg_kernel.hu"
__global__ void kernel0(int *grid)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 4097) {
      if (32 * b0 + t0 <= 2048) {
        grid[(32 * b0 + t0) * 2050 + 0] = grid[(32 * b0 + t0) * 2050 + 2048];
      } else if (32 * b0 + t0 >= 2050) {
        grid[(32 * b0 + t0 - 2049) * 2050 + 2049] = grid[(32 * b0 + t0 - 2049) * 2050 + 1];
      }
      if (b0 >= 64)
        grid[0 * 2050 + (32 * b0 + t0 - 2048)] = grid[2048 * 2050 + (32 * b0 + t0 - 2048)];
      if (32 * b0 + t0 <= 2050)
        grid[2049 * 2050 + (32 * b0 + t0 - 1)] = grid[1 * 2050 + (32 * b0 + t0 - 1)];
    }
}
__global__ void kernel1(int *grid, int *newGrid)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    int private_numNeighbors;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 2048)
      for (int c1 = ppcg_max(32768 * b0 + 32 * b1, 32 * ((b1 + 224) % 256) + 1024); c1 <= ppcg_min(2099200, 32768 * b0 + 33792); c1 += 8192)
        for (int c3 = ppcg_max(t1, ((t1 + c1 + 15) % 16) + 32768 * b0 + 1024 * t0 - c1 + 1); c3 <= ppcg_min(31, 32768 * b0 + 1024 * t0 - c1 + 2048); c3 += 16) {
          private_numNeighbors = (((((((grid[(32 * b0 + t0 + 1) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3)] + grid[(32 * b0 + t0 - 1) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3)]) + grid[(32 * b0 + t0) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3 + 1)]) + grid[(32 * b0 + t0) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3 - 1)]) + grid[(32 * b0 + t0 + 1) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3 + 1)]) + grid[(32 * b0 + t0 - 1) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3 - 1)]) + grid[(32 * b0 + t0 - 1) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3 + 1)]) + grid[(32 * b0 + t0 + 1) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3 - 1)]);
          if ((grid[(32 * b0 + t0) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3)] == 1) && (private_numNeighbors < 2)) {
            newGrid[(32 * b0 + t0) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3)] = 0;
          } else {
            if ((grid[(32 * b0 + t0) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3)] == 1) && ((private_numNeighbors == 2) || (private_numNeighbors == 3))) {
              newGrid[(32 * b0 + t0) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3)] = 1;
            } else {
              if ((grid[(32 * b0 + t0) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3)] == 1) && (private_numNeighbors > 3)) {
                newGrid[(32 * b0 + t0) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3)] = 0;
              } else {
                if ((grid[(32 * b0 + t0) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3)] == 0) && (private_numNeighbors == 3)) {
                  newGrid[(32 * b0 + t0) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3)] = 1;
                } else {
                  newGrid[(32 * b0 + t0) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3)] = grid[(32 * b0 + t0) * 2050 + (-32768 * b0 - 1024 * t0 + c1 + c3)];
                }
              }
            }
          }
        }
}
