
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
 
#define SRAND_VALUE 1985
#define BLOCK_SIZE 32

#define TIMESTEP 1024
#define SIDE 16384
#define CELL_TYPE int
#define CELL_NEIGHBOURS 8

// Classic GOL:
#define MIN_NOF_NEIGH_FROM_ALIVE_TO_ALIVE 2
#define MAX_NOF_NEIGH_FROM_ALIVE_TO_ALIVE 3
#define MIN_NOF_NEIGH_FROM_DEAD_TO_ALIVE 3
#define MAX_NOF_NEIGH_FROM_DEAD_TO_ALIVE 3

#define ALIVE  1
#define DEAD   0



__global__ void GOL(CELL_TYPE *d_grid, CELL_TYPE *d_newGrid)
{
    // We want id ∈ [1,dim]
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    //int id = i * SIDE + j;

    __shared__ CELL_TYPE rule_table [2][CELL_NEIGHBOURS+1];

    if ( threadIdx.y < 2 && threadIdx.x < (CELL_NEIGHBOURS+1) ) {
        // Init rule_table for GOL
        // Classic B3S23 GOL:
	    //rule_table[cases] = {
		//    d,d,d,a, d,d,d,d,d // DEAD is current state
		//    d,d,a,a, d,d,d,d,d // ALIVE is current state
		    if (threadIdx.y==0) 
                if (threadIdx.x >= MIN_NOF_NEIGH_FROM_DEAD_TO_ALIVE && threadIdx.x <= MAX_NOF_NEIGH_FROM_DEAD_TO_ALIVE)
			        rule_table[threadIdx.y][threadIdx.x] = ALIVE;
		        else
			        rule_table[threadIdx.y][threadIdx.x] = DEAD; 
 
		    if (threadIdx.y==1) 
                if (threadIdx.x >= MIN_NOF_NEIGH_FROM_ALIVE_TO_ALIVE && threadIdx.x <= MAX_NOF_NEIGH_FROM_ALIVE_TO_ALIVE)
			        rule_table[threadIdx.y][threadIdx.x] =  ALIVE;
		        else
			        rule_table[threadIdx.y][threadIdx.x] = DEAD;  
    }
    __syncthreads();


    if (i < SIDE && j < SIDE) {
        CELL_TYPE (*grid)[SIDE] = (CELL_TYPE (*)[SIDE]) d_grid;
        CELL_TYPE (*newGrid)[SIDE] = (CELL_TYPE (*)[SIDE]) d_newGrid;

 		int a1 = (i - 1 + SIDE)% SIDE;
		int a2 = (i + 1)% SIDE;
		int a3 = (j - 1 + SIDE)% SIDE;
		int a4 = (j + 1)% SIDE;
 
        // Get the number of neighbors for a given grid point
        newGrid[i][j] = rule_table [ grid[i][j] ] [ (grid[a1][j] + grid[a2][j] + grid[i][a3] + grid[i][a4] + grid[a1][a3] + grid[a1][a4] + grid[a2][a3] + grid[a2][a4]) ];
    }
}
 

int main(int argc, char* argv[])
{
    int i,j,iter;
    CELL_TYPE* h_grid; //Grid on host
    CELL_TYPE* d_grid; //Grid on device
    CELL_TYPE* d_newGrid; //Second grid used on device only
    CELL_TYPE* d_tmpGrid; //tmp grid pointer used to switch between grid and newGrid
  
    size_t bytes = sizeof(CELL_TYPE)*SIDE*SIDE;
    // Allocate host Grid used for initial setup and read back from device
    h_grid = (int*)malloc(bytes);
 
    // Allocate device grids
    hipMalloc(&d_grid, bytes);
    hipMalloc(&d_newGrid, bytes);
 
    // Assign initial population randomly
    srand(SRAND_VALUE);
    for(i = 0; i<SIDE; i++) {
        for(j = 0; j<SIDE; j++) {
            h_grid[i*SIDE+j] = rand() % 2;
        }
    }
 
    // Copy over initial game grid (Dim-1 threads)
    hipMemcpy(d_grid, h_grid, bytes, hipMemcpyHostToDevice);
 
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE,1);
    int linGrid = (int)ceil(SIDE/(float)BLOCK_SIZE);
    dim3 gridSize(linGrid,linGrid,1);
  

    // Main game loop
    for (iter = 0; iter<TIMESTEP; iter++) {
 
        GOL<<<gridSize, blockSize>>>(d_grid, d_newGrid);
 
        // Swap our grids and iterate again
        d_tmpGrid = d_grid;
        d_grid = d_newGrid;
        d_newGrid = d_tmpGrid;

    }//iter loop
 
    // Copy back results and sum
    hipMemcpy(h_grid, d_grid, bytes, hipMemcpyDeviceToHost);
 
    // Sum up alive cells and print results
    int total = 0;
    for (i = 0; i<SIDE; i++) {
        for (j = 0; j<SIDE; j++) {
            total += h_grid[i*SIDE+j];
        }
    }
    printf("Total Alive grid: %d\n", total);
 
    // Release memory
    hipFree(d_grid);
    hipFree(d_newGrid);
    free(h_grid);
 
    return 0;
}

// 256 Result in console: "Total Alive: 3281"
// 512 Result in console: "Total Alive: 11072"
// 1024 Result in console: "Total Alive: 45224"
// 2048 Result in console: "Total Alive: 182485"
// 4096 Result in console: "Total Alive: 724393"
// 8192 Result in console: "Total Alive: 2896683"
// 16384 Result in console: "Total Alive: 11547651"

