
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
 
#define SRAND_VALUE 1985
#define BLOCK_SIZE 32

#define TIMESTEP 1024
#define SIDE 16384
#define CELL_TYPE int
#define CELL_NEIGHBOURS 8

// Classic GOL:
#define MIN_NOF_NEIGH_FROM_ALIVE_TO_DEAD 2
#define MAX_NOF_NEIGH_FROM_ALIVE_TO_DEAD 3
#define MIN_NOF_NEIGH_FROM_DEAD_TO_ALIVE 3
#define MAX_NOF_NEIGH_FROM_DEAD_TO_ALIVE 3

#define ALIVE  1
#define DEAD   0

void print_matrix (CELL_TYPE *d_grid) {
    CELL_TYPE (*grid)[SIDE] = (CELL_TYPE (*)[SIDE]) d_grid;
    for(int i = 0; i<SIDE; i++) {
        for(int j = 0; j<SIDE; j++) {
            printf ("%d ", (int)grid[i][j]);
        }
            printf ("\n");
    }
}


__global__ void kernel_init_rule_table (CELL_TYPE *GPU_rule_table) {
    int i, idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    int value_alive, value_dead;

    // RULE COMPUTATION USING A TABLE . TABLE IS USED BELOW
	/* two simple examples
	    int d = DEAD, a = ALIVE;
    // Classic GOL:
	    rule_table[cases] = {
		    d,d,d,a, a,d,d,d, d, // DEAD is current state
		    d,d,d,d, a,a,d,d, d // ALIVE is current state
	    };
	*/
    for (i = idx; i < (CELL_NEIGHBOURS+1); i+= stride) {
		if (!(i >= MIN_NOF_NEIGH_FROM_ALIVE_TO_DEAD && i <= MAX_NOF_NEIGH_FROM_ALIVE_TO_DEAD))
			value_alive = DEAD;
		else
			value_alive = ALIVE;  

		if (i >= MIN_NOF_NEIGH_FROM_DEAD_TO_ALIVE && i <= MAX_NOF_NEIGH_FROM_DEAD_TO_ALIVE)
			value_dead = ALIVE;
		else
			value_dead = DEAD; 
 
		GPU_rule_table[i] = value_dead;
		GPU_rule_table[i + (CELL_NEIGHBOURS + 1)] = value_alive;

    }
}


__global__ void GOL(CELL_TYPE *d_grid, CELL_TYPE *d_newGrid, CELL_TYPE * GPU_rule_table)
{
    // We want id ∈ [1,dim]
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    //int id = iy * SIDE + ix;

    if (i < SIDE && j < SIDE) {
        CELL_TYPE (*grid)[SIDE] = (CELL_TYPE (*)[SIDE]) d_grid;
        CELL_TYPE (*newGrid)[SIDE] = (CELL_TYPE (*)[SIDE]) d_newGrid;
        CELL_TYPE (*rule_table)[CELL_NEIGHBOURS+1] = (CELL_TYPE (*)[CELL_NEIGHBOURS+1]) GPU_rule_table;
		int a1 = (i - 1 + SIDE)% SIDE;
		int a2 = (i + 1)% SIDE;
		int a3 = (j - 1 + SIDE)% SIDE;
		int a4 = (j + 1)% SIDE;
 
        // Get the number of neighbors for a given grid point
        newGrid[i][j] = rule_table [ grid[i][j] ] [ (grid[a1][j] + grid[a2][j] + grid[i][a3] + grid[i][a4] + grid[a1][a3] + grid[a1][a4] + grid[a2][a3] + grid[a2][a4]) ];
    }
}
 

int main(int argc, char* argv[])
{
    int i,j,iter;
    CELL_TYPE* h_grid; //Grid on host
    CELL_TYPE* d_grid; //Grid on device
    CELL_TYPE* d_newGrid; //Second grid used on device only
    CELL_TYPE* d_tmpGrid; //tmp grid pointer used to switch between grid and newGrid
    CELL_TYPE *GPU_rule_table;
  
    size_t bytes = sizeof(CELL_TYPE)*SIDE*SIDE;
    // Allocate host Grid used for initial setup and read back from device
    h_grid = (int*)malloc(bytes);
 
    // Allocate device grids
    hipMalloc(&d_grid, bytes);
    hipMalloc(&d_newGrid, bytes);
    hipMalloc(&GPU_rule_table, sizeof(CELL_TYPE)*2*(CELL_NEIGHBOURS+1));
 
    // Assign initial population randomly
    srand(SRAND_VALUE);
    for(i = 0; i<SIDE; i++) {
        for(j = 1; j<SIDE; j++) {
            h_grid[i*SIDE+j] = rand() % 2;
        }
    }
 
    // Copy over initial game grid (Dim-1 threads)
    hipMemcpy(d_grid, h_grid, bytes, hipMemcpyHostToDevice);
 
//print_matrix (h_grid);
//printf("\n");

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE,1);
    int linGrid = (int)ceil(SIDE/(float)BLOCK_SIZE);
    dim3 gridSize(linGrid,linGrid,1);
  
    kernel_init_rule_table<<<1,blockSize>>>(GPU_rule_table);

    // Main game loop
    for (iter = 0; iter<TIMESTEP; iter++) {
 
        GOL<<<gridSize, blockSize>>>(d_grid, d_newGrid, GPU_rule_table);

//cudaMemcpy(h_grid, d_newGrid, bytes, cudaMemcpyDeviceToHost);
//print_matrix (h_grid);
//printf("\n");
 
        // Swap our grids and iterate again
        d_tmpGrid = d_grid;
        d_grid = d_newGrid;
        d_newGrid = d_tmpGrid;

//cudaMemcpy(h_grid, d_grid, bytes, cudaMemcpyDeviceToHost);
//print_matrix (h_grid);
//printf("\n");


    }//iter loop
 
    // Copy back results and sum
    hipMemcpy(h_grid, d_grid, bytes, hipMemcpyDeviceToHost);
 
    // Sum up alive cells and print results
    int total = 0;
    for (i = 0; i<SIDE; i++) {
        for (j = 0; j<SIDE; j++) {
            total += h_grid[i*SIDE+j];
        }
    }
    printf("Total Alive grid: %d\n", total);
 
/*    cudaMemcpy(h_grid, d_newGrid, bytes, cudaMemcpyDeviceToHost);
 
    // Sum up alive cells and print results
    total=0;
    for (i = 0; i<SIDE; i++) {
        for (j = 0; j<SIDE; j++) {
            total += h_grid[i*SIDE+j];
        }
    }

    printf("Total Alive newGrid: %d\n", total);
*/

    // Release memory
    hipFree(d_grid);
    hipFree(d_newGrid);
    hipFree(GPU_rule_table);
    free(h_grid);
 
    return 0;
}
