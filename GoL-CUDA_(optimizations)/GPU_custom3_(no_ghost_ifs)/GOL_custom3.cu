
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
 
#define SRAND_VALUE 1985
#define BLOCK_SIZE 32

#define TIMESTEP 1024
#define SIDE 16384
#define CELL_TYPE int
#define CELL_NEIGHBOURS 8

 
__global__ void GOL(CELL_TYPE *d_grid, CELL_TYPE *d_newGrid)
{
    // We want id ∈ [1,dim]
    int iy = blockDim.y * blockIdx.y + threadIdx.y;
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int id = iy * SIDE + ix;
 
    if (iy < SIDE && ix < SIDE) {
        CELL_TYPE (*grid)[SIDE] = (CELL_TYPE (*)[SIDE]) d_grid;
        //CELL_TYPE (*newGrid)[SIDE] = (CELL_TYPE (*)[SIDE]) d_newGrid;
 		int a1 = (iy - 1 + SIDE)% SIDE;
		int a2 = (iy + 1)% SIDE;
		int a3 = (ix - 1 + SIDE)% SIDE;
		int a4 = (ix + 1)% SIDE;
 
        // Get the number of neighbors for a given grid point
        int numNeighbors = (grid[a1][ix] + grid[a2][ix] + grid[iy][a3] + grid[iy][a4] + grid[a1][a3] + grid[a1][a4] + grid[a2][a3] + grid[a2][a4]);
 
        CELL_TYPE cell = d_grid[id];
        // Here we have explicitly all of the game rules
        if (cell == 1 && numNeighbors < 2)
            d_newGrid[id] = 0;
        else if (cell == 1 && (numNeighbors == 2 || numNeighbors == 3))
            d_newGrid[id] = 1;
        else if (cell == 1 && numNeighbors > 3)
            d_newGrid[id] = 0;
        else if (cell == 0 && numNeighbors == 3)
            d_newGrid[id] = 1;
        else
            d_newGrid[id] = cell;
    }
}
 
int main(int argc, char* argv[])
{
    int i,j,iter;
    CELL_TYPE* h_grid; //Grid on host
    CELL_TYPE* d_grid; //Grid on device
    CELL_TYPE* d_newGrid; //Second grid used on device only
    CELL_TYPE* d_tmpGrid; //tmp grid pointer used to switch between grid and newGrid
  
    size_t bytes = sizeof(CELL_TYPE)*SIDE*SIDE;
    // Allocate host Grid used for initial setup and read back from device
    h_grid = (int*)malloc(bytes);
 
    // Allocate device grids
    hipMalloc(&d_grid, bytes);
    hipMalloc(&d_newGrid, bytes);
 
    // Assign initial population randomly
    srand(SRAND_VALUE);
    for(i = 0; i<SIDE; i++) {
        for(j = 0; j<SIDE; j++) {
            h_grid[i*SIDE+j] = rand() % 2;
        }
    }
 
    // Copy over initial game grid (Dim-1 threads)
    hipMemcpy(d_grid, h_grid, bytes, hipMemcpyHostToDevice);
 
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE,1);
    int linGrid = (int)ceil(SIDE/(float)BLOCK_SIZE);
    dim3 gridSize(linGrid,linGrid,1);
  

    // Main game loop
    for (iter = 0; iter<TIMESTEP; iter++) {
 
        GOL<<<gridSize, blockSize>>>(d_grid, d_newGrid);
 
        // Swap our grids and iterate again
        d_tmpGrid = d_grid;
        d_grid = d_newGrid;
        d_newGrid = d_tmpGrid;

    }//iter loop
 
    // Copy back results and sum
    hipMemcpy(h_grid, d_grid, bytes, hipMemcpyDeviceToHost);
 
    // Sum up alive cells and print results
    int total = 0;
    for (i = 0; i<SIDE; i++) {
        for (j = 0; j<SIDE; j++) {
            total += h_grid[i*SIDE+j];
        }
    }
    printf("Total Alive grid: %d\n", total);
 
    // Release memory
    hipFree(d_grid);
    hipFree(d_newGrid);
    free(h_grid);
 
    return 0;
}

// 256 Result in console: "Total Alive: 3281"
// 512 Result in console: "Total Alive: 11072"
// 1024 Result in console: "Total Alive: 45224"
// 2048 Result in console: "Total Alive: 182485"
// 4096 Result in console: "Total Alive: 724393"
// 8192 Result in console: "Total Alive: 2896683"

